#include "hip/hip_runtime.h"
/* gpmpn_divrem_1 -- mpn by limb division.

Copyright 1991, 1993, 1994, 1996, 1998-2000, 2002, 2003 Free Software
Foundation, Inc.

This file is part of the GNU MP Library.

The GNU MP Library is free software; you can redistribute it and/or modify
it under the terms of either:

  * the GNU Lesser General Public License as published by the Free
	Software Foundation; either version 3 of the License, or (at your
	option) any later version.

or

  * the GNU General Public License as published by the Free Software
	Foundation; either version 2 of the License, or (at your option) any
	later version.

or both in parallel, as here.

The GNU MP Library is distributed in the hope that it will be useful, but
WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
for more details.

You should have received copies of the GNU General Public License and the
GNU Lesser General Public License along with the GNU MP Library.  If not,
see https://www.gnu.org/licenses/.  */

#include "GPGMP/gpgmp-impl.cuh"
#include "GPGMP/longlong.cuh"

namespace gpgmp
{
	namespace mpnRoutines
	{

		/* The size where udiv_qrnnd_preinv should be used rather than udiv_qrnnd,
		meaning the quotient size where that should happen, the quotient size
		being how many udiv divisions will be done.

		The default is to use preinv always, CPUs where this doesn't suit have
		tuned thresholds.  Note in particular that preinv should certainly be
		used if that's the only division available (USE_PREINV_ALWAYS).  */

#ifndef DIVREM_1_NORM_THRESHOLD
#define DIVREM_1_NORM_THRESHOLD 0
#endif
#ifndef DIVREM_1_UNNORM_THRESHOLD
#define DIVREM_1_UNNORM_THRESHOLD 0
#endif

		//This serves as a direct wrapper for performing the udiv_qrnnd_preinv function.
		//This is done to avoid CUDA kernel launches erroring due to resource allocation issues when this macro is used inline in the gpmpn_divrem_1 function.
		ANYCALLER void perform_udiv_qrnnd_preinv(mp_limb_t& q, mp_limb_t& r, mp_limb_t& nh, mp_limb_t nl, mp_limb_t &d, mp_limb_t &dinv)
		{
			udiv_qrnnd_preinv(q, r, nh, nl, d, dinv);
		}


		/* If the cpu only has multiply-by-inverse division (eg. alpha), then NORM
		and UNNORM thresholds are 0 and only the inversion code is included.

		If multiply-by-inverse is never viable, then NORM and UNNORM thresholds
		will be MP_SIZE_T_MAX and only the plain division code is included.

		Otherwise mul-by-inverse is better than plain division above some
		threshold, and best results are obtained by having code for both present.

		The main reason for separating the norm and unnorm cases is that not all
		CPUs give zero for "n0 >> GMP_LIMB_BITS" which would arise in the unnorm
		code used on an already normalized divisor.

		If UDIV_NEEDS_NORMALIZATION is false then plain division uses the same
		non-shifting code for both the norm and unnorm cases, though with
		different criteria for skipping a division, and with different thresholds
		of course.  And in fact if inversion is never viable, then that simple
		non-shifting division would be all that's left.

		The NORM and UNNORM thresholds might not differ much, but if there's
		going to be separate code for norm and unnorm then it makes sense to have
		separate thresholds.  One thing that's possible is that the
		mul-by-inverse might be better only for normalized divisors, due to that
		case not needing variable bit shifts.

		Notice that the thresholds are tested after the decision to possibly skip
		one divide step, so they're based on the actual number of divisions done.

		For the unnorm case, it would be possible to call gpmpn_lshift to adjust
		the dividend all in one go (into the quotient space say), rather than
		limb-by-limb in the loop.  This might help if gpmpn_lshift is a lot faster
		than what the compiler can generate for EXTRACT.  But this is left to CPU
		specific implementations to consider, especially since EXTRACT isn't on
		the dependent chain.  */
		ANYCALLER mp_limb_t gpmpn_divrem_1(mp_ptr quotientStoreIn, mp_size_t fractionLimbs, mp_srcptr toDivide, mp_size_t numLimbsToDivide, mp_limb_t limbDivideBy)
		{
			mp_size_t n;
			mp_size_t i;
			mp_limb_t n1, n0;
			mp_limb_t r = 0;

			ASSERT(fractionLimbs >= 0);
			ASSERT(numLimbsToDivide >= 0);
			ASSERT(limbDivideBy != 0);
			/* FIXME: What's the correct overlap rule when qxn!=0? */
			ASSERT(MPN_SAME_OR_SEPARATE_P(quotientStoreIn + fractionLimbs, toDivide, numLimbsToDivide));

			n = numLimbsToDivide + fractionLimbs;
			if (n == 0)
				return 0;

				limbDivideBy <<= GMP_NAIL_BITS;

			quotientStoreIn += (n - 1); /* Make quotientStoreIn point at most significant quotient limb */

			if ((limbDivideBy & GMP_LIMB_HIGHBIT) != 0)
			{
				if (numLimbsToDivide != 0)
				{
					/* High quotient limb is 0 or 1, skip a divide step. */
					mp_limb_t q;
					r = toDivide[numLimbsToDivide - 1] << GMP_NAIL_BITS;
					q = (r >= limbDivideBy);
					*quotientStoreIn-- = q;
					r -= (limbDivideBy & -q);
					r >>= GMP_NAIL_BITS;
					n--;
					numLimbsToDivide--;
				}

				if (BELOW_THRESHOLD(n, DIVREM_1_NORM_THRESHOLD))
				{
				plain:
					for (i = numLimbsToDivide - 1; i >= 0; i--)
					{
						n0 = toDivide[i] << GMP_NAIL_BITS;
						udiv_qrnnd(*quotientStoreIn, r, r, n0, limbDivideBy);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					for (i = fractionLimbs - 1; i >= 0; i--)
					{
						udiv_qrnnd(*quotientStoreIn, r, r, CNST_LIMB(0), limbDivideBy);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					return r;
				}
				else
				{
					/* Multiply-by-inverse, divisor already normalized. */
					mp_limb_t dinv;
					invert_limb(dinv, limbDivideBy);

					for (i = numLimbsToDivide - 1; i >= 0; i--)
					{
						n0 = toDivide[i] << GMP_NAIL_BITS;
						udiv_qrnnd_preinv(*quotientStoreIn, r, r, n0, limbDivideBy, dinv);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					for (i = fractionLimbs - 1; i >= 0; i--)
					{
						perform_udiv_qrnnd_preinv(*quotientStoreIn, r, r, CNST_LIMB(0), limbDivideBy, dinv);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					return r;
				}
			}
			else
			{
				/* Most significant bit of divisor == 0.  */
				int cnt;

				/* Skip a division if high < divisor (high quotient 0).  Testing here
				before normalizing will still skip as often as possible.  */
				if (numLimbsToDivide != 0)
				{
					n1 = toDivide[numLimbsToDivide - 1] << GMP_NAIL_BITS;
					if (n1 < limbDivideBy)
					{
						r = n1 >> GMP_NAIL_BITS;
						*quotientStoreIn-- = 0;
						n--;
						if (n == 0)
							return r;
						numLimbsToDivide--;
					}
				}

				if (!UDIV_NEEDS_NORMALIZATION && BELOW_THRESHOLD(n, DIVREM_1_UNNORM_THRESHOLD))
					goto plain;

				count_leading_zeros(cnt, limbDivideBy);
				limbDivideBy <<= cnt;
				r <<= cnt;

				if (UDIV_NEEDS_NORMALIZATION && BELOW_THRESHOLD(n, DIVREM_1_UNNORM_THRESHOLD))
				{
					mp_limb_t nshift;
					if (numLimbsToDivide != 0)
					{
						n1 = toDivide[numLimbsToDivide - 1] << GMP_NAIL_BITS;
						r |= (n1 >> (GMP_LIMB_BITS - cnt));
						for (i = numLimbsToDivide - 2; i >= 0; i--)
						{
							n0 = toDivide[i] << GMP_NAIL_BITS;
							nshift = (n1 << cnt) | (n0 >> (GMP_NUMB_BITS - cnt));
							udiv_qrnnd(*quotientStoreIn, r, r, nshift, limbDivideBy);
							r >>= GMP_NAIL_BITS;
							quotientStoreIn--;
							n1 = n0;
						}
						udiv_qrnnd(*quotientStoreIn, r, r, n1 << cnt, limbDivideBy);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					for (i = fractionLimbs - 1; i >= 0; i--)
					{
						udiv_qrnnd(*quotientStoreIn, r, r, CNST_LIMB(0), limbDivideBy);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					return r >> cnt;
				}
				else
				{
					mp_limb_t dinv, nshift;
					invert_limb(dinv, limbDivideBy);
					if (numLimbsToDivide != 0)
					{
						n1 = toDivide[numLimbsToDivide - 1] << GMP_NAIL_BITS;
						r |= (n1 >> (GMP_LIMB_BITS - cnt));
						for (i = numLimbsToDivide - 2; i >= 0; i--)
						{
							n0 = toDivide[i] << GMP_NAIL_BITS;
							nshift = (n1 << cnt) | (n0 >> (GMP_NUMB_BITS - cnt));
							udiv_qrnnd_preinv(*quotientStoreIn, r, r, nshift, limbDivideBy, dinv);
							r >>= GMP_NAIL_BITS;
							quotientStoreIn--;
							n1 = n0;
						}
						udiv_qrnnd_preinv(*quotientStoreIn, r, r, n1 << cnt, limbDivideBy, dinv);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					for (i = fractionLimbs - 1; i >= 0; i--)
					{
						 udiv_qrnnd_preinv(*quotientStoreIn, r, r, CNST_LIMB(0), limbDivideBy, dinv);
						r >>= GMP_NAIL_BITS;
						quotientStoreIn--;
					}
					return r >> cnt;
				}
			}

			//This should never be reached, but is here to satisfy CUDA.
			return 0;
		}

	}
}