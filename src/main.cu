#include "hip/hip_runtime.h"
#include <iostream>
#include "gpgmp.cuh"

double ConvertBackToDouble(mpf_t val) {
    uint64_t intPart = 0;
    double fracPart = 0;

    //Important concepts:
    //  mp_d is the actual stored limb data inside of the mpf_t struct. It has a total size of mp_size limbs.
    //  The actual mp_d array can be conceptually split into two different arrays: an IntegerPart array and a FractionalPart array. FractionalPart comes first(if it exists! read on), IntegerPart comes second(if it exists).
    //  Both of these "sub-arrays" are in little-endian order, so the first limbs inside of them are their LEAST significant limbs.
    //  The mp_exp variable can be thought about as a number representing how many limbs to the LEFT the decimal point is. This means that FractionalPart's last index is at mp_size - mp_exp. MP_EXP CAN BE NEGATIVE! If it is, then the decimal point is even further to the right than the last limb by mp_exp limbs - this is used for values < 1 that are very, small in the fractional part - thus if mp_exp < 0 then there will never be an integer part.

    int integerArraySize = val->_mp_exp;
    int numLimbsUsed = ABS(val->_mp_size);
    int fractionalArraySize = numLimbsUsed - val->_mp_exp;

    //int fractionalArrayBeginningIdxInTotalArray = 0;
    int integerArrayBeginningIdxInTotalArray = numLimbsUsed - integerArraySize;

    mp_limb_t* fractionalSubArray = val->_mp_d; //simply mp_d since the fractional part comes first.
    mp_limb_t* integerSubArray = val->_mp_d + integerArrayBeginningIdxInTotalArray; //The integer array starts after the fractional array.


    //There's only an integer part to this value if integerArraySize(mp_exp) > 0 - otherwise, the entire array is a fractional part since the integer beginning being <=0 places to the left of the end of the array means that the entire array is a fractional part.
    if (integerArraySize > 0) {
        for (int intPartIdx = 0; intPartIdx < integerArraySize; intPartIdx++) {  // (val->_mp_size - (val->_mp_exp - 1)); intPartIdx < val->_mp_size; intPartIdx++) {
            intPart += integerSubArray[intPartIdx] * pow(2, GMP_NUMB_BITS * (intPartIdx)); //integerSubArray[intPartIdx] w/o simplified variable names expands to val->_mp_d[(val->_mp_size - val->_mp_exp) + intPartIdx]
        }
    }

    for (int fracPartIdx = 0; fracPartIdx < fractionalArraySize; fracPartIdx++) {
        mp_limb_t fracPartLimb = fractionalSubArray[fracPartIdx];
        double doubleFracPart = static_cast<double>(fracPartLimb);

        /*
            This variable is the "size contribution" of the current fractional limb;
            it is the amount of times which we need to shift the limb to the right by <bitsPerLimb>(i.e limb / 2^(GMP_NUMB_BITS*sizeContrib)) in order to transform the limb to...
            ...the correct decimal size to be able to add the limb's value to a double correctly.

            We decrement fracPartIdx from (mp_size - mp_exp) since:
              Ontop of this little bit of array concatenation confusion, we also need to keep in mind that each of these "sub-arrays" are in little-endian order, therefore to find the true "size contribution" of any limb we actually need...
              ...to invert the index relative to the end of its sub-array! So (endOfSubArray - currentIndexFromStartOfSubArray) will give us the true size contribution of any limb;
            Therefore (val->_mp_size(totalSizeOfBothArrays) - val->_mp_exp(sizeOfIntegerArray) is the size of FractionalPart; then we subtract fracPartIdx from that size to get the "size contribution"
        */
        int invertedFractionalLimbIndexStartingFromLastIndexInFractionalArray = fractionalArraySize - fracPartIdx;
        //We now have what we would 'normally' think of as the index of the current fractional limb in the fractional limb array, as if it were big endian.
        //This is useful because it represents how far the limb is from the right of the decimal point; specifically it means that we are <limbBits>*n bits to the right of the decimal point

        //...We can use that index as a multiplier for the division's power.
        int bitsToTheRightOfDecimalPointThatThisLimbRepresents = GMP_NUMB_BITS * invertedFractionalLimbIndexStartingFromLastIndexInFractionalArray;

        //This double is the actual value of the current fractional limb, converted to a double.
        double fracPartRepresentedByThisLimb = (doubleFracPart / (pow(2, bitsToTheRightOfDecimalPointThatThisLimbRepresents)));
        fracPart += fracPartRepresentedByThisLimb;
    }

    return (fracPart + static_cast<double>(intPart)) * SGN(val->_mp_size);
}

ANYCALLER void PrintDataAboutMPNArray(gpgmp::mpn_array* array) {
    printf("Array data(%d integers in array, %d limbs allocated per integer):\n", array->numIntegersInArray, array->numLimbsPerInteger);
    for (int i = 0; i < array->numIntegersInArray; i++) {
        printf("    Integer #%d", i);
        if (MPN_ARRAY_SIZES(array)[i] < 0) {
            printf("(NEGATIVE)");
        }
        printf(":\n");
        for (int limbIdx = 0; limbIdx < array->numLimbsPerInteger; limbIdx++) {
            printf("        Limb #%d: %llu\n", limbIdx, MPN_ARRAY_DATA(array)[i * array->numLimbsPerInteger + limbIdx]);
        }
    }
}

__global__ void testKernel(gpgmp::mpn_device_array deviceArray) {
    printf("Hello, world! I'm a kernel with a thread index of %d!\n", threadIdx.x);

    PrintDataAboutMPNArray(deviceArray);
}

#define CHECK_CUDA_ERROR(err) do { \
    if (err != hipSuccess) { \
        printf("CUDA error: %s\nError detected during check on line %d\n", hipGetErrorString(err), __LINE__); \
        exit(1); \
    } \
} while (0)


#define NUM_INTEGERS_IN_ARRAY 5
#define PRECISION_PER_INTEGER 64*2
int main(int argc, char** argv) {
    __gmpf_set_default_prec(64*5);
    hipError_t err;

    gpgmp::mpn_device_array testArray;
    printf("Allocating array on GPU...\n");

    err = gpgmp::host::mpn_array_allocate_on_device(testArray, NUM_INTEGERS_IN_ARRAY, PRECISION_PER_INTEGER);
    CHECK_CUDA_ERROR(err);

    printf("Array allocated!\n");

    printf("Initializing array on GPU from CPU...\n");
    mpz_t mpzArray[NUM_INTEGERS_IN_ARRAY];
    for (int i = 0; i < NUM_INTEGERS_IN_ARRAY; i++) {
        mpz_init_set_si(mpzArray[i], i);
    }
    mpz_init_set_d(mpzArray[0], UINT64_MAX);
    mpz_init_set_d(mpzArray[1], UINT64_MAX);
    mpz_init_set_d(mpzArray[2], UINT64_MAX);

    err = gpgmp::host::mpn_array_init_on_device_from_mpz_array(testArray, mpzArray, NUM_INTEGERS_IN_ARRAY, PRECISION_PER_INTEGER, 5);
    CHECK_CUDA_ERROR(err);
    printf("Array initialized!\n");

    printf("Launching kernel...\n");

    testKernel<<<1, 1>>>(testArray);
    hipDeviceSynchronize();
    printf("Kernel finished!\n");


    printf("Testing custom mpn_add_n...\n");
    mpz_t mpzResult, mpzTest1, mpzTest2;
    mpz_init(mpzResult);
    mpz_init(mpzTest1);
    mpz_init(mpzTest2);
    mpz_set_d(mpzResult, UINT64_MAX);
    mpz_set_d(mpzTest1, UINT64_MAX);
    mpz_set_d(mpzTest2, UINT64_MAX);

    mp_srcptr operand1_ptr = mpzTest1->_mp_d;
    mp_srcptr operand2_ptr = mpzTest2->_mp_d;
    mp_size_t size = mpzTest1->_mp_size;

    mp_limb_t carry = gpgmp::mpnRoutines::mpn_add_n(mpzResult->_mp_d, operand1_ptr, operand2_ptr, size);
    printf("(Final Carry = %llu)\n", carry);

    printf("(CPU) mpzTest1 = (%llu*(2^64)) + %llu\n", mpzTest1->_mp_d[1], mpzTest1->_mp_d[0]);
    printf("(CPU) mpzTest2 = (%llu*(2^64)) + %llu\n", mpzTest2->_mp_d[1], mpzTest2->_mp_d[0]);
    printf("(CPU) Result = (%llu*(2^64)) + %llu\n", mpzResult->_mp_d[1], mpzResult->_mp_d[0]);


    return 0;
}