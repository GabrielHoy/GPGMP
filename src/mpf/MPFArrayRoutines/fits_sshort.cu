#include "hip/hip_runtime.h"
#include "gpgmp-impl.cuh"

namespace gpgmp
{
  namespace mpfArrayRoutines
  {

    ANYCALLER int
    mpf_fits_sshort_p(mpf_array_idx f) __GMP_NOTHROW
    {
      mp_size_t fs, fn;
      mp_srcptr fp;
      mp_exp_t exp;
      mp_limb_t fl;

      exp = MPF_ARRAY_EXPONENTS(f.array)[f.idx];
      if (exp < 1)
        return 1;

      fs = MPF_ARRAY_SIZES(f.array)[f.idx];
      fp = MPF_ARRAY_DATA_AT_IDX(f.array, f.idx);
      fn = ABS(fs);

      if (exp == 1)
      {
        fl = fp[fn - 1];
      }
#if GMP_NAIL_BITS != 0
      else if (exp == 2 && SHRT_MAX > GMP_NUMB_MAX)
      {
        fl = fp[fn - 1];
        if ((fl >> GMP_NAIL_BITS) != 0)
          return 0;
        fl = (fl << GMP_NUMB_BITS);
        if (fn >= 2)
          fl |= fp[fn - 2];
      }
#endif
      else
        return 0;

      return fl <= (fs >= 0 ? static_cast<mp_limb_t>(SHRT_MAX) : NEG_CAST(mp_limb_t, SHRT_MIN));
    }

  }
}