#include "hip/hip_runtime.h"
#include "gpgmp-impl.cuh"

namespace gpgmp
{
  namespace mpfArrayRoutines
  {

    /* Multiples of GMP_NUMB_BITS in exp simply mean an amount added to EXP(u)
       to set EXP(r).  The remainder exp%GMP_NUMB_BITS is then a left shift for
       the limb data.

       If exp%GMP_NUMB_BITS == 0 then there's no shifting, we effectively just
       do an mpz_set with changed EXP(r).  Like mpz_set we take prec+1 limbs in
       this case.  Although just prec would suffice, it's nice to have
       mpf_mul_2exp with exp==0 come out the same as mpz_set.

       When shifting we take up to prec many limbs from the input.  Our shift is
       cy = mpn_lshift (PTR(r), PTR(u)+k, size, ...), where k is the number of
       low limbs dropped from u, and the carry out is stored to PTR(r)[size].

       It may be noted that the low limb PTR(r)[0] doesn't incorporate bits from
       PTR(u)[k-1] (when k>=1 makes that limb available).  Taking just prec
       limbs from the input (with the high non-zero) is enough bits for the
       application requested precision, there's no need for extra work.

       If r==u the shift will have overlapping operands.  When k==0 (ie. when
       usize <= prec), the overlap is supported by lshift (ie. dst == src).

       But when r==u and k>=1 (ie. usize > prec), we would have an invalid
       overlap (ie. mpn_lshift (rp, rp+k, ...)).  In this case we must instead
       use mpn_rshift (PTR(r)+1, PTR(u)+k, size, NUMB-shift) with the carry out
       stored to PTR(r)[0].  An rshift by NUMB-shift bits like this gives
       identical data, it's just its overlap restrictions which differ.

       Enhancements:

       The way mpn_lshift is used means successive mpf_mul_2exp calls on the
       same operand will accumulate low zero limbs, until prec+1 limbs is
       reached.  This is wasteful for subsequent operations.  When abs_usize <=
       prec, we should test the low exp%GMP_NUMB_BITS many bits of PTR(u)[0],
       ie. those which would be shifted out by an mpn_rshift.  If they're zero
       then use that mpn_rshift.  */

    ANYCALLER void
    gpmpf_mul_2exp(mpf_array_idx r, mpf_array_idx u, mp_bitcnt_t exp)
    {
      mp_srcptr up;
      mp_ptr rp = MPF_ARRAY_DATA_AT_IDX(r.array, r.idx);
      mp_size_t usize;
      mp_size_t abs_usize;
      mp_size_t prec = r.array->userSpecifiedPrecisionLimbCount;
      mp_exp_t uexp = MPF_ARRAY_EXPONENTS(u.array)[u.idx];

      usize = MPF_ARRAY_SIZES(u.array)[u.idx];

      if (UNLIKELY(usize == 0))
      {
        MPF_ARRAY_SIZES(r.array)[r.idx] = 0;
        MPF_ARRAY_EXPONENTS(r.array)[r.idx] = 0;
        return;
      }

      abs_usize = ABS(usize);
      up = MPF_ARRAY_DATA_AT_IDX(u.array, u.idx);

      if (exp % GMP_NUMB_BITS == 0)
      {
        prec++; /* retain more precision here as we don't need
         to account for carry-out here */
        if (abs_usize > prec)
        {
          up += abs_usize - prec;
          abs_usize = prec;
        }
        if (rp != up)
          MPN_COPY_INCR(rp, up, abs_usize);
        MPF_ARRAY_EXPONENTS(r.array)[r.idx] = uexp + exp / GMP_NUMB_BITS;
      }
      else
      {
        mp_limb_t cy_limb;
        mp_size_t adj;
        if (abs_usize > prec)
        {
          up += abs_usize - prec;
          abs_usize = prec;
          /* Use mpn_rshift since mpn_lshift operates downwards, and we
             therefore would clobber part of U before using that part, in case
             R is the same variable as U.  */
          cy_limb = gpgmp::mpnRoutines::gpmpn_rshift(rp + 1, up, abs_usize,
                               GMP_NUMB_BITS - exp % GMP_NUMB_BITS);
          rp[0] = cy_limb;
          adj = rp[abs_usize] != 0;
        }
        else
        {
          cy_limb = gpgmp::mpnRoutines::gpmpn_lshift(rp, up, abs_usize, exp % GMP_NUMB_BITS);
          rp[abs_usize] = cy_limb;
          adj = cy_limb != 0;
        }

        abs_usize += adj;
        MPF_ARRAY_EXPONENTS(r.array)[r.idx] = uexp + exp / GMP_NUMB_BITS + adj;
      }
      MPF_ARRAY_SIZES(r.array)[r.idx] = usize >= 0 ? abs_usize : -abs_usize;
    }

  }
}