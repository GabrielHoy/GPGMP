#include "hip/hip_runtime.h"
#include "gpgmp-impl.cuh"

namespace gpgmp
{
  namespace mpfArrayRoutines
  {

    /* As usual the aim is to produce PREC(r) limbs, with the high non-zero.  The
       basic mpn_div_q produces a quotient of nsize-dsize+1 limbs, with either the
       high or second highest limb non-zero.  We arrange for nsize-dsize+1 to equal
       prec+1, hence giving either prec or prec+1 result limbs at PTR(r).

       nsize-dsize+1 == prec+1 is achieved by adjusting num(q), either dropping low
       limbs if it's too big, or padding with low zeros if it's too small.  The
       full given den(q) is always used.

       We cannot truncate den(q), because even when it's much bigger than prec the
       last limbs can still influence the final quotient.  Often they don't, but we
       leave optimization of that to mpn_div_q.

       Enhancements:

       The high quotient limb is non-zero when high{np,dsize} > {dp,dsize}.  We
       could make that comparison and use qsize==prec instead of qsize==prec+1,
       to save one limb in the division.  */

    //TODO: Refactor to avoid dynamic allocation.
    ANYCALLER void
    gpmpf_set_q(mpf_array_idx r, mpq_srcptr q)
    {
      mp_srcptr np, dp;
      mp_size_t prec, nsize, dsize, qsize, prospective_qsize, tsize, zeros;
      mp_size_t sign_quotient, high_zero;
      mp_ptr qp, tp;
      mp_exp_t exp;
      TMP_DECL;

      ASSERT(SIZ(&q->_mp_den) > 0); /* canonical q */

      nsize = SIZ(&q->_mp_num);
      dsize = SIZ(&q->_mp_den);

      if (UNLIKELY(nsize == 0))
      {
        MPF_ARRAY_SIZES(r.array)[r.idx] = 0;
        MPF_ARRAY_EXPONENTS(r.array)[r.idx] = 0;
        return;
      }

      TMP_MARK;

      prec = r.array->userSpecifiedPrecisionLimbCount;
      qp = MPF_ARRAY_DATA_AT_IDX(r.array, r.idx);

      sign_quotient = nsize;
      nsize = ABS(nsize);
      np = PTR(&q->_mp_num);
      dp = PTR(&q->_mp_den);

      prospective_qsize = nsize - dsize + 1; /* q from using given n,d sizes */
      exp = prospective_qsize;               /* ie. number of integer limbs */
      qsize = prec + 1;                      /* desired q */

      zeros = qsize - prospective_qsize; /* n zeros to get desired qsize */
      tsize = nsize + zeros;             /* size of intermediate numerator */
      tp = TMP_ALLOC_LIMBS(tsize + 1);   /* +1 for mpn_div_q's scratch */

      if (zeros > 0)
      {
        /* pad n with zeros into temporary space */
        MPN_ZERO(tp, zeros);
        MPN_COPY(tp + zeros, np, nsize);
        np = tp; /* mpn_div_q allows this overlap */
      }
      else
      {
        /* shorten n to get desired qsize */
        np -= zeros;
      }

      ASSERT(tsize - dsize + 1 == qsize);
      gpgmp::mpnRoutines::gpmpn_div_q(qp, np, tsize, dp, dsize, tp);

      /* strip possible zero high limb */
      high_zero = (qp[qsize - 1] == 0);
      qsize -= high_zero;
      exp -= high_zero;

      MPF_ARRAY_EXPONENTS(r.array)[r.idx] = exp;
      MPF_ARRAY_SIZES(r.array)[r.idx] = sign_quotient >= 0 ? qsize : -qsize;

      TMP_FREE;
    }

  }
}